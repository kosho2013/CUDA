#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>

using namespace std;

#define TILE_SIZE 1024
#define STEPS 10

__global__ void vecAdd(float *in, float *out)
{
    int start = blockIdx.x * TILE_SIZE;
    int tx = threadIdx.x;

    __shared__ float tile[TILE_SIZE];

    // loading
    tile[tx] = in[start + tx];
    __syncthreads();
    
    // computation
    int stride = 1;
    for (int i = 0; i < STEPS; i++)
    {
      if (tx >= stride)
      {
        float tmp;
        tmp = tile[tx] + tile[tx - stride];
        __syncthreads();
        tile[tx] = tmp;
        __syncthreads();
      }
      stride *= 2;
    }

    out[start + tx] = tile[tx];
}

int main(int argc, char **argv) {
  size_t size_in = TILE_SIZE * sizeof(float);
  size_t size_out = TILE_SIZE * sizeof(float);

  // Allocate and initialize host memory
  float *h_in = (float *)malloc(2 * size_in);
  float *h_out = (float *)malloc(size_out);

  for (int i = 0; i < TILE_SIZE; i++)
  {
    h_in[i] = i; 
  }

  // Allocate device memory
  float *d_in, *d_out;
  hipMalloc((void **)&d_in, size_in);
  hipMalloc((void **)&d_out, size_out);

  // Copy data from host to device
  hipMemcpy(d_in, h_in, size_in, hipMemcpyHostToDevice);

  // Launch the kernel
  dim3 blockDim(TILE_SIZE);
  dim3 gridDim(1);

  // Create CUDA event to measure time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start);

  vecAdd<<<gridDim, blockDim>>>(d_in, d_out);

  // Record the stop event
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Calculate elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Copy result back to host
  hipMemcpy(h_out, d_out, size_out, hipMemcpyDeviceToHost);


  // Print the elapsed time
  std::cout << "blockDim: (" << blockDim.x << ", " << blockDim.y << ", " << blockDim.z << ")" << std::endl;
  std::cout << "gridDim: (" << gridDim.x << ", " << gridDim.y << ", " << gridDim.z << ")" << std::endl;
  std::cout << "Elapsed time: " << milliseconds << " ms" << std::endl;
  


  float *gold = (float *)malloc(size_out);
  float value = 0.0f;
  for (int i = 0; i < TILE_SIZE; i++)
  {
    value += h_in[i];
    gold[i] = value;
  }

  for (int i = 0; i < TILE_SIZE; i++)
  {
    float err = fabs(gold[i] - h_out[i]);
    if (err > 0.01)
    {
      cout << i << " " << gold[i] << " " << h_out[i] << endl;
    }
  }

  // Free device memory
  hipFree(d_in);
  hipFree(d_out);

  // Free host memory
  free(h_in);
  free(h_out);

  return 0;
}
