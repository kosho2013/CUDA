#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;


#define TILE 1024


__global__ void kernel(float *A, float *B, float *C, const int m, const int k, const int n)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < m)
    {
      float dot = 0;
      for (int y = 0; y < k; ++y)
      {
          dot += A[y * k + x] * B[y];
      }
      C[x] = dot; 

    }
    
}


int main(int argc, char **argv) {
  int m = 10000;
  int k = 10000;
  int n = 1;

  size_t A_size = m * k * sizeof(float);
  size_t B_size = k * n * sizeof(float);
  size_t C_size = m * n * sizeof(float);
  
  // Allocate and initialize host memory
  float *h_A = (float *)malloc(A_size);
  float *h_B = (float *)malloc(B_size);
  float *h_C = (float *)malloc(C_size);

  for (int i = 0; i < m * k; i++) {
    h_A[i] = (i + 22) % 44;
  }

  for (int i = 0; i < k * n; i++) {
    h_B[i] = (i - 22) % 64;
  }



  // Allocate device memory
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, A_size);
  hipMalloc((void **)&d_B, B_size);
  hipMalloc((void **)&d_C, C_size);

  // Copy data from host to device
  hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);
  

  // Create CUDA event to measure time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start);


  dim3 blockDim(TILE);
  int numBlocks = (m + TILE - 1) / TILE;
  dim3 gridDim(numBlocks);
  kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, k, n);

  // Record the stop event
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Calculate elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Copy result back to host
  hipMemcpy(h_C, d_C, C_size, hipMemcpyDeviceToHost);


  // Print the elapsed time
  std::cout << "blockDim: (" << blockDim.x << ", " << blockDim.y << ", " << blockDim.z << ")" << std::endl;
  std::cout << "gridDim: (" << gridDim.x << ", " << gridDim.y << ", " << gridDim.z << ")" << std::endl;
  std::cout << "Elapsed time: " << milliseconds << " ms" << std::endl;
  
  float *gold = (float *)malloc(C_size);
  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      float value = 0.0f;
      for (int v = 0; v < k; v++)
      {
        value += h_A[i*k+v] * h_B[v*n+j];
      }
      gold[i*n+j] = value;
    }
  }


//   for (int i = 0; i < m*n; i++)
//   {
//       float err = fabs(gold[i] - h_C[i]);
//       if (err > 0.1)
//       {
//         cout << err << endl;
//       }
//   }

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
