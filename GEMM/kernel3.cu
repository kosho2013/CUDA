#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;


#define TILE 32

__global__ void kernel(float *A, float *B, float *C, int m, int k, int n)
{


    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float A_tile[TILE][TILE];
    __shared__ float B_tile[TILE][TILE];


    int steps = (k + TILE - 1) / TILE;
    float value = 0.0f;

    for (int i = 0; i < steps; i++)
    {
        if (x < m && i*TILE+ty < k)
        {
            A_tile[tx][ty] = A[x * k + i*TILE+ty];
        } else
        {
            A_tile[tx][ty] = 0.0f;
        }


        if (y < n && i*TILE+tx < k)
        {
            B_tile[tx][ty] = B[(i*TILE+tx) * n + y];
        } else
        {
            B_tile[tx][ty] = 0.0f;
        }

        __syncthreads();

        for (int j = 0; j < TILE; j++)
        {
            value += A_tile[tx][j] * B_tile[j][ty];
        }

        __syncthreads();
    }

    if (x < m && y < n)
    {
        C[x * n + y] = value;
    }
}



int main(int argc, char **argv) {
  int m = 1000;
  int k = 1000;
  int n = 1000;

  size_t A_size = m * k * sizeof(float);
  size_t B_size = k * n * sizeof(float);
  size_t C_size = m * n * sizeof(float);
  
  // Allocate and initialize host memory
  float *h_A = (float *)malloc(A_size);
  float *h_B = (float *)malloc(B_size);
  float *h_C = (float *)malloc(C_size);

  for (int i = 0; i < m * k; i++) {
    h_A[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  for (int i = 0; i < k * n; i++) {
    h_B[i] = static_cast<float>(rand()) / RAND_MAX;
  }



  // Allocate device memory
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, A_size);
  hipMalloc((void **)&d_B, B_size);
  hipMalloc((void **)&d_C, C_size);

  // Copy data from host to device
  hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);
  

  // Create CUDA event to measure time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start);


  dim3 blockDim(TILE, TILE);
  int a = (m + TILE - 1) / TILE;
  int b = (n + TILE - 1) / TILE;
  dim3 gridDim(a, b);
  kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, k, n);

  // Record the stop event
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Calculate elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Copy result back to host
  hipMemcpy(h_C, d_C, C_size, hipMemcpyDeviceToHost);


  // Print the elapsed time
  std::cout << "blockDim: (" << blockDim.x << ", " << blockDim.y << ", " << blockDim.z << ")" << std::endl;
  std::cout << "gridDim: (" << gridDim.x << ", " << gridDim.y << ", " << gridDim.z << ")" << std::endl;
  std::cout << "Elapsed time: " << milliseconds << " ms" << std::endl;
  
  float *gold = (float *)malloc(C_size);
  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      float value = 0.0f;
      for (int v = 0; v < k; v++)
      {
        value += h_A[i*k+v] * h_B[v*n+j];
      }
      gold[i*n+j] = value;
    }
  }


  for (int i = 0; i < m*n; i++)
  {
      float err = fabs(gold[i] - h_C[i]);
      if (err > 0.1)
      {
        cout << err << endl;
      }
  }

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
