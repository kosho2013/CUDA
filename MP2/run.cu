#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void vecAdd(float *A, float *B, float *C, int m, int k, int n) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x < m && y < n)
  {
    float value = 0.0f;
    for (int i = 0; i < k; ++i)
    {
      value += A[x*k + i] * B[i*n + y];
    }
    C[x*n + y] = value;
  }
}

int main(int argc, char **argv) {
  int m = 100;
  int n = 200;
  int k = 300;

  size_t A_size = m * k * sizeof(float);
  size_t B_size = k * n * sizeof(float);
  size_t C_size = m * n * sizeof(float);
  
  // Allocate and initialize host memory
  float *h_A = (float *)malloc(A_size);
  float *h_B = (float *)malloc(B_size);
  float *h_C = (float *)malloc(C_size);

  for (int i = 0; i < m*k; i++) {
    h_A[i] = 5.0f;
  }

  for (int i = 0; i < k*n; i++) {
    h_B[i] = 5.0f;
  }

  // Allocate device memory
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, A_size);
  hipMalloc((void **)&d_B, B_size);
  hipMalloc((void **)&d_C, C_size);

  // Copy data from host to device
  hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);

  // Launch the kernel
  int block_1 = 32;
  int block_2 = 32;
  int numBlock_1 = (m + block_1 - 1) / block_1;
  int numBlock_2 = (n + block_2 - 1) / block_2;

  dim3 blockDim(block_1, block_2);
  dim3 gridDim(numBlock_1, numBlock_2);

  // Create CUDA event to measure time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start);

  vecAdd<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, k, n);

  // Record the stop event
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Calculate elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Copy result back to host
  hipMemcpy(h_C, d_C, C_size, hipMemcpyDeviceToHost);

  // Print the elapsed time
  std::cout << "blockDim: (" << blockDim.x << ", " << blockDim.y << ", " << blockDim.z << ")" << std::endl;
  std::cout << "gridDim: (" << gridDim.x << ", " << gridDim.y << ", " << gridDim.z << ")" << std::endl;
  std::cout << "Elapsed time: " << milliseconds << " ms" << std::endl;
  
  float *gold = (float *)malloc(C_size);
  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      float value = 0.0f;
      for (int v = 0; v < k; v++)
      {
        value += h_A[i*k+v] * h_B[v*n+j];
      }
      gold[i*n+j] = value;
    }
  }


  for (int i = 0; i < m*n; i++)
  {
      float err = fabs(gold[i] - h_C[i]);
      if (err > 0.01)
      {
        cout << err << endl;
      }
  }


  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
